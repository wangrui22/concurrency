#include "hip/hip_runtime.h"
#ifdef WIN32
#include "../include/gl/glew.h"
#include "../include/gl/freeglut.h"
#else 
#include "GL/glew.h"
#include "GL/freeglut.h"
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>  
#include <cuda_gl_interop.h>
#include <hip/hip_texture_types.h>
#include <hip/hip_vector_types.h>
#include <iostream>
#include <fstream>


#define CHECK_CUDA_ERROR {\
hipError_t err = hipGetLastError(); \
if (err != hipSuccess) {\
    std::cout << "CUDA error: " << err << " in function: " << __FUNCTION__ <<\
    " line: " << __LINE__ << std::endl; \
}}\

static int _width = 1024;
static int _height = 1024;
static unsigned char* _checkboard_data = nullptr;
static unsigned char* _checkboard_data_host = nullptr;
texture<uchar4, 2, hipReadModeElementType> texRef;

void create_data() {
    _checkboard_data = new unsigned char[_width*_height * 4];
    int tag_x = 0;
    int tag_y = 0;
    int idx = 0;
    for (int y = 0; y < _height; ++y) {
        for (int x = 0; x < _width; ++x) {
            tag_x = x / 32;
            tag_y = y / 32;
            idx = y*_width + x;
            if ((tag_x + tag_y) % 2 == 0) {
                _checkboard_data[idx * 4] = 200;
                _checkboard_data[idx * 4 + 1] = 200;
                _checkboard_data[idx * 4 + 2] = 200;
                _checkboard_data[idx * 4 + 3] = 255;
            }
            else {
                _checkboard_data[idx * 4] = 20;
                _checkboard_data[idx * 4 + 1] = 20;
                _checkboard_data[idx * 4 + 2] = 20;
                _checkboard_data[idx * 4 + 3] = 255;
            }
        }
    }
}

__global__ void tansfromKernel(unsigned char* output, int width, int height) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    float u = x / (float)width;
    float v = y / (float)height;

    uchar4 rgba = tex2D(texRef, x, y);
    int idx = y*width + x;
    output[idx * 4] = (unsigned char)(rgba.x);
    output[idx * 4 + 1] = 20;//  change R 
    output[idx * 4 + 2] = (unsigned char)(rgba.z);
    output[idx * 4 + 3] = 255;
}

int cuda_texture(int argc, char* argv[]) {
    create_data();

    //CUDA array
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(
        8, 8, 8, 8, hipChannelFormatKindUnsigned);
    hipArray* cuda_array;
    hipMallocArray(&cuda_array, &channel_desc, _width, _height);
    
    CHECK_CUDA_ERROR;

    //copy data to CUDA array
    hipMemcpyToArray(cuda_array, 0, 0, _checkboard_data, _width*_height*4, hipMemcpyHostToDevice);
    hipBindTextureToArray(&texRef, cuda_array, &channel_desc);

    CHECK_CUDA_ERROR;

    ////Cuda resource
    //struct hipResourceDesc  res_desc;
    //memset(&res_desc, 0, sizeof(hipResourceDesc));
    //res_desc.resType = hipResourceTypeArray;
    //res_desc.res.array.array = cuda_array;
    //
    ////Texture parameter (like GL's glTexParameteri)
    //struct hipTextureDesc tex_desc;
    //memset(&tex_desc,0, sizeof(hipTextureDesc));
    //tex_desc.addressMode[0] = hipAddressModeWrap;
    //tex_desc.addressMode[1] = hipAddressModeWrap;
    //tex_desc.filterMode = hipFilterModeLinear;
    //tex_desc.readMode = hipReadModeNormalizedFloat;
    //tex_desc.normalizedCoords = 1;

    ////create texture
    //hipTextureObject_t tex_obj = 0;
    //hipCreateTextureObject(&tex_obj, &res_desc, &tex_desc, NULL);

    CHECK_CUDA_ERROR;

    unsigned char* output = nullptr;
    hipMalloc(&output,_width*_height*sizeof(unsigned char)*4);
   
    CHECK_CUDA_ERROR;

    //invoke
#define BLOCK_SIZE 16
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(_width/ BLOCK_SIZE, _height/ BLOCK_SIZE);
    tansfromKernel<<<grid,block>>>(output, _width, _height);

    hipUnbindTexture(&texRef);

    hipDeviceSynchronize();
    CHECK_CUDA_ERROR;

    _checkboard_data_host = new unsigned char[_width*_height*4];
    hipMemcpy(_checkboard_data_host, output, _width*_height*4, hipMemcpyDefault);
    
    CHECK_CUDA_ERROR; 

    std::ofstream out("D:/temp/tex_res.raw", std::ios::out | std::ios::binary );
    if (out.is_open()) {
        out.write((char*)_checkboard_data_host, _width*_height*4);
        out.close();
        std::cout << "write done.";
    }
    std::cout << "done.";

    return 0;
}


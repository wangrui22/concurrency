#include "hip/hip_runtime.h"
#ifdef WIN32
#include "../include/gl/glew.h"
#include "../include/gl/freeglut.h"
#else 
#include "GL/glew.h"
#include "GL/freeglut.h"
#endif

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>  
#include <cuda_gl_interop.h>
#include <hip/hip_texture_types.h>
#include <hip/hip_vector_types.h>
#include <iostream>
#include <fstream>

static int _win_width = 1024;
static int _win_height = 1024;
static GLuint _tex_id;
static GLuint _tex_id_const;
static unsigned char* _tex_buffer = nullptr;
hipGraphicsResource* _cuda_gl_resource = nullptr;
hipArray *_cuda_array = nullptr;
texture<uchar3, hipTextureType2D, hipReadModeElementType> _tex_ref;

void gl_init() {
    _tex_buffer = new unsigned char[_win_width*_win_height*3];
    int tag_x = 0;
    int tag_y = 0;
    int idx = 0;
    for (int y = 0; y < _win_height; ++y) {
        for (int x = 0; x < _win_width; ++x) {
            tag_x = x/32;
            tag_y = y/32;
            idx = y*_win_width + x;
            if ((tag_x + tag_y) % 2 == 0) {
                _tex_buffer[idx*3] = 200;
                _tex_buffer[idx * 3+1] = 200;
                _tex_buffer[idx * 3+2] = 200;
            }
            else {
                _tex_buffer[idx * 3] = 20;
                _tex_buffer[idx * 3 + 1] = 20;
                _tex_buffer[idx * 3 + 2] = 20;
            }

        }
    }
    
    {
        //debug
        /*std::ofstream out("D:/temp/tex.raw",std::ios::out|std::ios::binary);
        if (out.is_open()) {
            out.write((char*)_tex_buffer, _win_width*_win_height*3);
        }
        out.close();*/
    }
    
    glEnable(GL_TEXTURE_2D);
    
    glGenTextures(1, &_tex_id_const);
    glBindTexture(GL_TEXTURE_2D, _tex_id_const);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB8, _win_width, _win_height, 0, GL_RGB, GL_UNSIGNED_BYTE, _tex_buffer);

    glGenTextures(1, &_tex_id);
    glBindTexture(GL_TEXTURE_2D, _tex_id);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB8, _win_width, _win_height, 0, GL_RGB, GL_UNSIGNED_BYTE, _tex_buffer);


    

}

void cuda_init() {
    glBindTexture(GL_TEXTURE_2D, _tex_id);
    hipError_t err = hipGraphicsGLRegisterImage(&_cuda_gl_resource, _tex_id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
    if (err != hipSuccess) {
        std::cout << "regtister GL image failed.\n";
        return;
    }
    
    //when shutdown tht application . should call hipGraphicsUnregisterResource
}


//read const checkboard change some value , write to new texture(PBO)
__global__ update_checkboard() {
    
}   

void cuda_update_checkboard() {
    hipGraphicsMapResources(1, &_cuda_gl_resource, 0);
    hipGraphicsSubResourceGetMappedArray(&_cuda_array, _cuda_gl_resource, 0,0);
    hipBindTextureToArray(tex_ref, (hipArray*)_cuda_array);

    //launch kernel

    hipGraphicsUnmapResources(1,&_cuda_gl_resource,0);

    //test
}

void cuda_use


void display() {
    glClearColor(0,0,0,0);
    glClearDepth(0.0);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    
    glPushMatrix();
    glPushAttrib(GL_ALL_ATTRIB_BITS);

    glDisable(GL_BLEND);
    glDepthMask(GL_FALSE);
    glDisable(GL_DEPTH_TEST);


    glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    //glColor3f(1.0f,0.0f,0.0f);
    glBindTexture(GL_TEXTURE_2D, _tex_id);
    glBegin(GL_QUADS);
    glTexCoord2f(0,0);
    glVertex2d(-1.0, -1.0);
    glTexCoord2f(1, 0);
    glVertex2d(1.0, -1.0);
    glTexCoord2f(1, 1);
    glVertex2d(1.0, 1.0);
    glTexCoord2f(0, 1);
    glVertex2d(-1.0, 1.0);
    glEnd();
    
    glPopAttrib();
    glPopMatrix();
    
    glutSwapBuffers();
}


int cuda_gl(int argc, char* argv[]) {
    
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_DOUBLE | GL_RGB);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(_win_width,_win_height);
    glutCreateWindow("cuda GL");
    
    if (GLEW_OK != glewInit()) {
        std::cout << "Init glew failed!\n";
        return -1;
    }

    gl_init();
    cuda_init();

    glutDisplayFunc(display);
    glutMainLoop();

    return 0;
}
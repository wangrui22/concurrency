#include "hip/hip_runtime.h"
#include <stdlib.h>  
#include <stdio.h>  
#include <hip/hip_runtime.h>  
#include <hip/hip_runtime.h>  
#include <assert.h>    


__global__ void cu_arrayDelete(int* arrayIO)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    arrayIO[idx] = arrayIO[idx] - 16;
}
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        printf("Cuda error: %s: %s./n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
int page_locked_mem(int argc, char *argv[])
{
    int* h_pData = NULL;
    int* d_pData = NULL;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    if (!deviceProp.canMapHostMemory) {
        printf("Device %d cannot map host memory!/n");
    }
    hipSetDeviceFlags(hipDeviceMapHost);
    checkCUDAError("hipSetDeviceFlags");

    hipHostAlloc(&h_pData, 512, hipHostMallocMapped);
    hipHostGetDevicePointer((void **)&d_pData, (void *)h_pData, 0);
    for (int i = 0; i<128; i++)
    {
        h_pData[i] = 255;
    }
    cu_arrayDelete << <4, 32 >> >(d_pData);
    hipDeviceSynchronize();
    for (int i = 0; i<128; i++)
        printf("%d/n", h_pData[0]);
    return 0;
}
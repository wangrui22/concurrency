#include "hip/hip_runtime.h"

#include "hip/driver_types.h"
#include <stdio.h>
#include <fstream>

#define BLOCK_SIZE 16

typedef struct
{
    int width;
    int height;
    float* elements;
} Matrix;

__global__ void mat_mul_kernel(const Matrix a, const Matrix b, Matrix c) {
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    float res = 0.0;
    for (int i = 0; i < a.width; ++i) {
        res += a.elements[row*a.height + i] * b.elements[i*b.width + col];
    }
    c.elements[row*c.width + col] = res;
}

int mat_mul(const Matrix a, const Matrix b, Matrix c) 
{
    //1 copy host memory to device(cudaMalloc + cudaMemcpy)
    hipError_t cuda_error = hipSuccess;
    Matrix d_a;
    d_a.width = a.width;
    d_a.height = a.height;
    cuda_error = hipMalloc(&(d_a.elements), d_a.width*d_a.height*sizeof(float));
    if (cuda_error != hipSuccess) {
        printf("cuda malloc failed." );
        return -1;
    }
    cuda_error = hipMemcpy(d_a.elements, a.elements, d_a.width*d_a.height * sizeof(float), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        printf("cuda memcpy failed.");
        return -1;
    }

    Matrix d_b;
    d_b.width = b.width;
    d_b.height = b.height;
    cuda_error = hipMalloc(&(d_b.elements), d_b.width*d_b.height * sizeof(float));
    if (cuda_error != hipSuccess) {
        printf("cuda malloc failed.");
        return -1;
    }
    cuda_error = hipMemcpy(d_b.elements, b.elements, d_b.width*d_b.height * sizeof(float), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        printf("cuda memcpy failed.");
        return -1;
    }

    Matrix d_c;
    d_c.width = c.width;
    d_c.height = c.height;
    cuda_error = hipMalloc(&(d_c.elements), d_c.width*d_c.height * sizeof(float));
    if (cuda_error != hipSuccess) {
        printf("cuda malloc failed.");
        return -1;
    }

    //2 invoke kernel to calculate (block thread)
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(b.width / dimBlock.x, a.height / dimBlock.y);
    //dim3 dimGrid = (4, 4);
    mat_mul_kernel<<<dimGrid,dimBlock>>>(d_a, d_b, d_c);

    //3 download to host
    hipMemcpy(c.elements, d_c.elements, c.width*c.height*sizeof(float) , hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess) {
        printf("cuda memcpy failed.");
        return -1;
    }

    //4 release the device memeory
    hipFree(d_a.elements);
    hipFree(d_b.elements);
    hipFree(d_c.elements);

    return 0;
}

int save_matrix(Matrix m, const char* file_name) {
    std::ofstream out(file_name, std::ios::out);
    if (!out.is_open()) {
        return -1;
    }

    for (int row = 0; row < m.height; ++row) {
        for (int col= 0; col < m.width; ++col) {
            out << m.elements[row*m.height + col] << " ";
        }
        out << std::endl;
    }

    out.close();
    return 0;
}

int cuda_matrix_mul(int argc , char* argv[]) 
{
    Matrix a;
    a.width = 64;
    a.height = 64;
    a.elements = new float[64*64];
    for (int i = 0; i < 64 * 64; ++i) {
        a.elements[i] = 1.0f;
    }

    Matrix b;
    b.width = 64;
    b.height = 64;
    b.elements = new float[64 * 64];
    for (int i = 0; i < 64 * 64; ++i) {
        b.elements[i] = 2.0f;
    }

    Matrix c;
    c.width = 64;
    c.height = 64;
    c.elements = new float[64 * 64];
    for (int i = 0; i < 64 * 64; ++i) {
        c.elements[i] = 0;
    }

    if (0 == mat_mul(a, b, c)) {
        printf("Success.\n");
        save_matrix(c, "D:/temp/mat.txt");
    }
    else {
        printf("Failed.\n");
    }
    
    return 0;
}